/*
 * HogGPU.cpp
 *
 *  Created on: May 12, 2015
 *      Author: marcelo
 */

#include <include/HogGPU.h>
#include "HogGPU_impl.cuh"

#include <iostream>

#include <boost/thread.hpp>

#include <opencv2/gpu/gpu.hpp>

#include <include/Utils.h>

namespace ghog
{
namespace lib
{

HogGPU::HogGPU(std::string settings_file) :
	HogDescriptor(settings_file)
{

}

HogGPU::~HogGPU()
{
// TODO Auto-generated destructor stub
}

void HogGPU::alloc_buffer(cv::Size buffer_size,
	int type,
	cv::Mat& buffer)
{
	cv::gpu::CudaMem cudamem(buffer_size.height, buffer_size.width, type,
		cv::gpu::CudaMem::ALLOC_ZEROCOPY);
	buffer = cudamem.createMatHeader();
	buffer.refcount = cudamem.refcount;
	buffer.addref();
	buffer.setTo(0);
}

GHOG_LIB_STATUS HogGPU::image_normalization(cv::Mat& image,
	ImageCallback* callback)
{
	boost::thread(&HogGPU::image_normalization_async, this, image, callback)
		.detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::image_normalization_sync(cv::Mat& image)
{
	//TODO
}

GHOG_LIB_STATUS HogGPU::calc_gradient(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase,
	GradientCallback* callback)
{
	boost::thread(&HogGPU::calc_gradient_async, this, input_img, magnitude,
		phase, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::calc_gradient_sync(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase)
{
	dim3 block_size(8, 8);
	dim3 grid_size;
	grid_size.x = input_img.cols / block_size.x;
	grid_size.y = input_img.rows / block_size.y;

	if(input_img.cols % block_size.x)
	{
		grid_size.x++;
	}
	if(input_img.rows % block_size.y)
	{
		grid_size.y++;
	}

	float* input_img_ptr = input_img.ptr< float >(0);
	float* magnitude_ptr = magnitude.ptr< float >(0);
	float* phase_ptr = phase.ptr< float >(0);

	float* device_input_img;
	float* device_magnitude;
	float* device_phase;

	hipHostGetDevicePointer(&device_input_img, input_img_ptr, 0);
	hipHostGetDevicePointer(&device_magnitude, magnitude_ptr, 0);
	hipHostGetDevicePointer(&device_phase, phase_ptr, 0);

	gradient_kernel<<<grid_size, block_size>>>(device_input_img,
		device_magnitude, device_phase, input_img.rows, input_img.cols,
		input_img.step1(), magnitude.step1(), phase.step1());
	hipDeviceSynchronize();
}

GHOG_LIB_STATUS HogGPU::create_descriptor(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat& descriptor,
	DescriptorCallback* callback)
{
	boost::thread(&HogGPU::create_descriptor_async, this, magnitude, phase,
		descriptor, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::create_descriptor_sync(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat& descriptor)
{
	//TODO: verify that magnitude and phase have correct size and type.
	//TODO: verify that the descriptor has correct size and type
	//TODO: possibly preallocate histograms auxiliary matrix

	cv::Size hog_block_grid(
		((_cell_grid.width - _block_size.width) / _block_stride.width) + 1,
		((_cell_grid.height - _block_size.height) / _block_stride.height) + 1);

	dim3 block_size(8, 8);
	dim3 grid_size;
	grid_size.x = _cell_grid.width / block_size.x;
	grid_size.y = _cell_grid.height / block_size.y;

	if(_cell_grid.width % block_size.x)
	{
		grid_size.x++;
	}
	if(_cell_grid.height % block_size.y)
	{
		grid_size.y++;
	}

	float* magnitude_ptr = magnitude.ptr< float >(0);
	float* phase_ptr = phase.ptr< float >(0);
	float* descriptor_ptr = descriptor.ptr< float >(0);

	float* device_magnitude;
	float* device_phase;
	float* device_descriptor;

	hipHostGetDevicePointer(&device_magnitude, magnitude_ptr, 0);
	hipHostGetDevicePointer(&device_phase, phase_ptr, 0);
	hipHostGetDevicePointer(&device_descriptor, descriptor_ptr, 0);

	float* device_histograms;
	int cell_row_step = _cell_grid.width * _num_bins;

	hipMalloc((void**)&device_histograms,
		(_cell_grid.height * cell_row_step * sizeof(float)));

	histogram_kernel<<<grid_size, block_size>>>(device_magnitude, device_phase,
		device_histograms, _cell_grid.width, _cell_grid.height,
		magnitude.step1(), phase.step1(), cell_row_step, _cell_size.width,
		_cell_size.height, _num_bins);

	float * histograms;
	histograms = (float*)malloc(
		_cell_grid.height * cell_row_step * sizeof(float));
	hipMemcpy(histograms, device_histograms,
		(_cell_grid.height * cell_row_step * sizeof(float)),
		hipMemcpyDeviceToHost);

	for(int i = 0; i < _cell_grid.height * cell_row_step; ++i)
	{
		if(std::fpclassify(histograms[i]) == FP_NAN)
		{
			std::cout << "NAN on histograms[" << i << "]" << std::endl;
		}
	}

	grid_size.x = _cell_grid.width / block_size.x;
	grid_size.y = _cell_grid.height / block_size.y;

	if(_cell_grid.width % block_size.x)
	{
		grid_size.x++;
	}
	if(_cell_grid.height % block_size.y)
	{
		grid_size.y++;
	}

	hipDeviceSynchronize();
	block_normalization_kernel<<<grid_size, block_size>>>(device_histograms,
		device_descriptor, hog_block_grid.width, hog_block_grid.height,
		_block_size.width, _block_size.height, _num_bins, _cell_grid.width,
		_block_stride.width, _block_stride.height);
	hipDeviceSynchronize();
}

} /* namespace lib */
} /* namespace ghog */

