#include "hip/hip_runtime.h"
/*
 * HogGPU.cpp
 *
 *  Created on: May 12, 2015
 *      Author: marcelo
 */

#include <include/HogGPU.h>
#include "HogGPU_impl.cuh"

#include <iostream>

#include <boost/thread.hpp>

#include <opencv2/gpu/gpu.hpp>

#include <include/Utils.h>

namespace ghog
{
namespace lib
{

HogGPU::HogGPU(std::string settings_file) :
	_settings(settings_file)
{
	_classifier = NULL;

	load_settings(settings_file);
}

HogGPU::~HogGPU()
{
// TODO Auto-generated destructor stub
}

void HogGPU::alloc_buffer(cv::Size buffer_size,
	int type,
	cv::Mat& buffer)
{
	cv::gpu::CudaMem cudamem(buffer_size.height, buffer_size.width, type,
		cv::gpu::CudaMem::ALLOC_ZEROCOPY);
	buffer = cudamem.createMatHeader();
	buffer.refcount = cudamem.refcount;
	buffer.addref();
	buffer.setTo(0);
}

GHOG_LIB_STATUS HogGPU::image_normalization(cv::Mat& image,
	ImageCallback* callback)
{
	boost::thread(&HogGPU::image_normalization_async, this, image, callback)
		.detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::image_normalization_async(cv::Mat& image,
	ImageCallback* callback)
{
	image_normalization_sync(image);
	callback->image_processed(image);
}

void HogGPU::image_normalization_sync(cv::Mat& image)
{
	//TODO
}

GHOG_LIB_STATUS HogGPU::calc_gradient(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase,
	GradientCallback* callback)
{
	boost::thread(&HogGPU::calc_gradient_async, this, input_img, magnitude,
		phase, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::calc_gradient_async(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase,
	GradientCallback* callback)
{
	calc_gradient_sync(input_img, magnitude, phase);
	callback->gradients_obtained(magnitude, phase);
}

void HogGPU::calc_gradient_sync(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase)
{
	dim3 block_size(8, 8);
	dim3 grid_size;
	grid_size.x = input_img.cols / block_size.x;
	grid_size.y = input_img.rows / block_size.y;

	if(input_img.cols % block_size.x)
	{
		grid_size.x++;
	}
	if(input_img.rows % block_size.y)
	{
		grid_size.y++;
	}

	float* input_img_ptr = input_img.ptr< float >(0);
	float* magnitude_ptr = magnitude.ptr< float >(0);
	float* phase_ptr = phase.ptr< float >(0);

	float* device_input_img;
	float* device_magnitude;
	float* device_phase;

	hipHostGetDevicePointer(&device_input_img, input_img_ptr, 0);
	hipHostGetDevicePointer(&device_magnitude, magnitude_ptr, 0);
	hipHostGetDevicePointer(&device_phase, phase_ptr, 0);

	gradient_kernel<<<grid_size, block_size>>>(device_input_img,
		device_magnitude, device_phase, input_img.rows, input_img.cols,
		input_img.step1(), magnitude.step1(), phase.step1());
	hipDeviceSynchronize();
}

GHOG_LIB_STATUS HogGPU::create_descriptor(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat& descriptor,
	DescriptorCallback* callback)
{
	boost::thread(&HogGPU::create_descriptor_async, this, magnitude, phase,
		descriptor, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::create_descriptor_async(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat& descriptor,
	DescriptorCallback* callback)
{
	create_descriptor_sync(magnitude, phase, descriptor);
	callback->descriptor_obtained(descriptor);
}

void HogGPU::create_descriptor_sync(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat& descriptor)
{
	//TODO: verify that magnitude and phase have same size and type.
	//TODO: get preallocated descriptor and verify it, instead of creating.

	cv::Size cell_grid = Utils::partition(magnitude.size(), _cell_size);
	int total_cells = cell_grid.width * cell_grid.height;
	int blocks_per_cell = _block_size.width * _block_size.height;
	int total_outputs = total_cells * blocks_per_cell * _num_bins;
	descriptor.create(1, total_outputs, CV_32FC1);
	cv::Mat histograms(total_cells, _num_bins, CV_32FC1);
	int top_row = 0, bottom_row = 0, left_col = 0, right_col = 0;
	int extra_rows = magnitude.rows % cell_grid.height;
	int extra_cols = magnitude.cols % cell_grid.width;
	int output_col_left = 0;
	int output_col_right = _num_bins;
	int histogram_row = 0;

	for(int i = 0; i < cell_grid.height; ++i)
	{
		bottom_row = top_row += _cell_size.height;
		if(extra_rows > 0)
		{
			extra_rows--;
			bottom_row++;
		}
		cv::Mat mag_aux = magnitude.rowRange(top_row, bottom_row);
		cv::Mat phase_aux = phase.rowRange(top_row, bottom_row);
		for(int j = 0; j < cell_grid.width; ++j)
		{
			right_col = left_col += _cell_size.width;
			if(extra_cols > 0)
			{
				extra_cols--;
				right_col++;
			}
			calc_histogram(mag_aux.colRange(left_col, right_col),
				phase_aux.colRange(left_col, right_col),
				histograms.row(histogram_row));
			histogram_row++;
			left_col = right_col;
		}
		extra_cols = magnitude.cols % cell_grid.width;
		top_row = bottom_row;
	}

	//TODO: normalize and put on the descriptor
}

void HogGPU::calc_histogram(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat histogram)
{
	float bin_size = 360.0f / (float)_num_bins;

	int left_bin, right_bin;
	float delta;

	float mag_total = 0.0f;

	for(int i = 0; i < magnitude.rows; ++i)
	{
		for(int j = 0; j < magnitude.cols; ++j)
		{
			if(magnitude.at< float >(i, j) > 0)
			{
				left_bin = (int)floor(
					(phase.at< float >(i, j) - bin_size / 2.0f) / bin_size);
				if(left_bin < 0)
					left_bin += _num_bins;
				right_bin = (left_bin + 1) % _num_bins;

				delta = (phase.at< float >(i, j) / bin_size) - right_bin;
				if(right_bin == 0)
					delta -= _num_bins;

				histogram.at< float >(left_bin) += (0.5 - delta)
					* magnitude.at< float >(i, j);
				histogram.at< float >(right_bin) += (0.5 + delta)
					* magnitude.at< float >(i, j);
				mag_total += magnitude.at< float >(i, j);
			}
		}
	}

	for(int i = 0; i < _num_bins; ++i)
	{
		histogram.at< float >(i) /= mag_total;
	}
}

GHOG_LIB_STATUS HogGPU::classify(cv::Mat img,
	ClassifyCallback* callback)
{
	boost::thread(&HogGPU::classify_async, this, img, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::classify_async(cv::Mat img,
	ClassifyCallback* callback)
{
	callback->classification_result(classify_sync(img));
}

bool HogGPU::classify_sync(cv::Mat img)
{
	bool ret = false;
	cv::Mat resized;
	image_normalization_sync(img);
	cv::Mat grad_mag;
	cv::Mat grad_phase;
	calc_gradient_sync(img, grad_mag, grad_phase);
	cv::Mat descriptor;
	create_descriptor_sync(grad_mag, grad_phase, descriptor);
	cv::Mat output = _classifier->classify_sync(descriptor);
	if(output.at< float >(0) > 0)
	{
		ret = true;
	}
	return ret;
}

GHOG_LIB_STATUS HogGPU::locate(cv::Mat img,
	cv::Rect roi,
	cv::Size window_size,
	cv::Size window_stride,
	LocateCallback* callback)
{
	boost::thread(&HogGPU::locate_async, this, img, roi, window_size,
		window_stride, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::locate_async(cv::Mat img,
	cv::Rect roi,
	cv::Size window_size,
	cv::Size window_stride,
	LocateCallback* callback)
{
	callback->objects_located(
		locate_sync(img, roi, window_size, window_stride));
}

std::vector< cv::Rect > HogGPU::locate_sync(cv::Mat img,
	cv::Rect roi,
	cv::Size window_size,
	cv::Size window_stride)
{
	std::vector< cv::Rect > ret;
	return ret;
}

void HogGPU::load_settings(std::string filename)
{
	_num_bins = _settings.load_int(std::string("Descriptor"), "NUMBER_OF_BINS");
	_block_size.width = _settings.load_int(std::string("Descriptor"),
		"BLOCK_SIZE_COLS");
	_block_size.height = _settings.load_int(std::string("Descriptor"),
		"BLOCK_SIZE_ROWS");
	_cell_size.width = _settings.load_int(std::string("Descriptor"),
		"CELL_SIZE_COLS");
	_cell_size.height = _settings.load_int(std::string("Descriptor"),
		"CELL_SIZE_ROWS");
}

void HogGPU::set_classifier(IClassifier* classifier)
{
	_classifier = classifier;
}

GHOG_LIB_STATUS HogGPU::set_param(std::string param,
	std::string value)
{
	std::string module = get_module(param);
	if(module == "NULL")
	{
		return GHOG_LIB_STATUS_INVALID_PARAMETER_NAME;
	}
	_settings.save(module, param, value.c_str());
	return GHOG_LIB_STATUS_OK;
}

std::string HogGPU::get_param(std::string param)
{
	std::string module = get_module(param);
	if(module == "NULL")
	{
		return "Invalid parameter name.";
	} else
	{
		return _settings.load_str(module, param);
	}
}

std::string HogGPU::get_module(std::string param_name)
{
	if((param_name == "CELL_SIZE_COLS") || (param_name == "CELL_SIZE_ROWS")
		|| (param_name == "BLOCK_SIZE_COLS")
		|| (param_name == "BLOCK_SIZE_ROWS")
		|| (param_name == "NUMBER_OF_BINS"))
	{
		return "Descriptor";
	} else if((param_name == "TYPE") || (param_name == "FILENAME"))
	{
		return "Classifier";
	} else
	{
		return "NULL";
	}
}

} /* namespace lib */
} /* namespace ghog */

