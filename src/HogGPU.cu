#include "hip/hip_runtime.h"
/*
 * HogGPU.cpp
 *
 *  Created on: May 12, 2015
 *      Author: marcelo
 */

#include <include/HogGPU.h>
#include "HogGPU_impl.cuh"

#include <iostream>

#include <boost/thread.hpp>

#include <opencv2/gpu/gpu.hpp>

namespace ghog
{
namespace lib
{

HogGPU::HogGPU(std::string settings_file) :
	_settings(settings_file)
{
	_classifier = NULL;

	load_settings(settings_file);
}

HogGPU::~HogGPU()
{
// TODO Auto-generated destructor stub
}

void HogGPU::alloc_buffer(cv::Size buffer_size,
	int type,
	cv::Mat& buffer)
{
	cv::gpu::CudaMem cudamem(buffer, cv::gpu::CudaMem::ALLOC_ZEROCOPY);

///////////////////////////////////////////////////////////////////////////////
	std::cout << "Ref counter -> cmem: ";
	if(cudamem.refcount != 0)
	{
		std::cout << (*(cudamem.refcount)) << " @ " << cudamem.refcount;
	} else
	{
		std::cout << " invalid @ 0x00";
	}
	std::cout << "  mat: ";
	if(buffer.refcount != 0)
	{
		std::cout << (*(buffer.refcount)) << " @ " << buffer.refcount;
	} else
	{
		std::cout << " invalid @ 0x00";
	}
	std::cout << std::endl;
///////////////////////////////////////////////////////////////////////////////

	cudamem.create(buffer_size.height, buffer_size.width, type);
	buffer = cudamem.createMatHeader();

///////////////////////////////////////////////////////////////////////////////
	if(cudamem.refcount != 0)
	{
		std::cout << (*(cudamem.refcount)) << " @ " << cudamem.refcount;
	} else
	{
		std::cout << " invalid @ 0x00";
	}
	std::cout << "  mat: ";
	if(buffer.refcount != 0)
	{
		std::cout << (*(buffer.refcount)) << " @ " << buffer.refcount;
	} else
	{
		std::cout << " invalid @ 0x00";
	}
///////////////////////////////////////////////////////////////////////////////

	buffer.addref();

///////////////////////////////////////////////////////////////////////////////
	if(cudamem.refcount != 0)
	{
		std::cout << (*(cudamem.refcount)) << " @ " << cudamem.refcount;
	} else
	{
		std::cout << " invalid @ 0x00";
	}
	std::cout << "  mat: ";
	if(buffer.refcount != 0)
	{
		std::cout << (*(buffer.refcount)) << " @ " << buffer.refcount;
	} else
	{
		std::cout << " invalid @ 0x00";
	}
///////////////////////////////////////////////////////////////////////////////

}

GHOG_LIB_STATUS HogGPU::resize(cv::Mat image,
	cv::Size new_size,
	cv::Mat& resized_image,
	ImageCallback* callback)
{
	boost::thread(&HogGPU::resize_async, this, image, new_size, resized_image,
		callback).detach();
	return GHOG_LIB_STATUS_OK;
}

GHOG_LIB_STATUS HogGPU::calc_gradient(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase,
	GradientCallback* callback)
{
	boost::thread(&HogGPU::calc_gradient_async, this, input_img, magnitude,
		phase, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

GHOG_LIB_STATUS HogGPU::create_descriptor(cv::Mat gradients,
	cv::Size block_size,
	int num_bins,
	DescriptorCallback* callback)
{
	boost::thread(&HogGPU::create_descriptor_async, this, gradients, block_size,
		num_bins, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

GHOG_LIB_STATUS HogGPU::classify(cv::Mat img,
	ClassifyCallback* callback)
{
	boost::thread(&HogGPU::classify_async, this, img, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

GHOG_LIB_STATUS HogGPU::locate(cv::Mat img,
	cv::Rect roi,
	cv::Size window_size,
	cv::Size window_stride,
	LocateCallback* callback)
{
	boost::thread(&HogGPU::locate_async, this, img, roi, window_size,
		window_stride, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::load_settings(std::string filename)
{
	_img_resize.width = _settings.load_int("Hog",
		"CLASSIFICATION_IMAGE_HEIGHT");
	_img_resize.width = _settings.load_int("Hog", "CLASSIFICATION_IMAGE_WIDTH");

	_num_bins = _settings.load_int(std::string("Descriptor"), "NUMBER_OF_BINS");
	_block_size.width = _settings.load_int(std::string("Descriptor"),
		"BLOCK_SIZE_COLS");
	_block_size.height = _settings.load_int(std::string("Descriptor"),
		"BLOCK_SIZE_ROWS");
}

void HogGPU::set_classifier(IClassifier* classifier)
{
	_classifier = classifier;
}

GHOG_LIB_STATUS HogGPU::set_img_resize(cv::Size img_resize)
{
	_img_resize = img_resize;
	return GHOG_LIB_STATUS_OK;
}

cv::Size HogGPU::get_img_resize()
{
	return _img_resize;
}

GHOG_LIB_STATUS HogGPU::set_num_bins(int num_bins)
{
	_num_bins = num_bins;
	return GHOG_LIB_STATUS_OK;
}

int HogGPU::get_num_bins()
{
	return _num_bins;
}

GHOG_LIB_STATUS HogGPU::set_block_size(cv::Size block_size)
{
	_block_size = block_size;
	return GHOG_LIB_STATUS_OK;
}

cv::Size HogGPU::get_block_size()
{
	return _block_size;
}

void HogGPU::resize_async(cv::Mat image,
	cv::Size new_size,
	cv::Mat& resized_image,
	ImageCallback* callback)
{
	resize_impl(image, new_size, resized_image);
	callback->image_processed(image, resized_image);
}

void HogGPU::calc_gradient_async(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase,
	GradientCallback* callback)
{
	calc_gradient_impl(input_img, magnitude, phase);
	callback->gradients_obtained(input_img, magnitude, phase);
}

void HogGPU::create_descriptor_async(cv::Mat gradients,
	cv::Size block_size,
	int num_bins,
	DescriptorCallback* callback)
{
	cv::Mat ret;
	create_descriptor_impl(gradients, block_size, num_bins, ret);
	callback->descriptor_obtained(gradients, ret);
}

void HogGPU::classify_async(cv::Mat img,
	ClassifyCallback* callback)
{
	bool ret = false;
	cv::Mat resized;
	resize_impl(img, _img_resize, resized);
	cv::Mat grad_mag;
	cv::Mat grad_phase;
	calc_gradient_impl(img, grad_mag, grad_phase);
	cv::Mat descriptor;
	create_descriptor_impl(resized, _block_size, _num_bins, descriptor);
	cv::Mat output = _classifier->classify_sync(descriptor);
	if(output.at< float >(0) > 0)
	{
		ret = true;
	}
	callback->classification_result(img, ret);
}

void HogGPU::locate_async(cv::Mat img,
	cv::Rect roi,
	cv::Size window_size,
	cv::Size window_stride,
	LocateCallback* callback)
{
	std::vector< cv::Rect > ret;
	callback->objects_located(img, ret);
}

void HogGPU::resize_impl(cv::Mat image,
	cv::Size new_size,
	cv::Mat& resized_image)
{
	cv::gpu::CudaMem input(image, cv::gpu::CudaMem::ALLOC_ZEROCOPY);
	cv::gpu::CudaMem output(resized_image, cv::gpu::CudaMem::ALLOC_ZEROCOPY);
	cv::gpu::GpuMat output_gpu = output.createGpuMatHeader();
	cv::gpu::resize(input.createGpuMatHeader(), output_gpu, new_size, 0, 0,
		CV_INTER_LINEAR);
}

void HogGPU::calc_gradient_impl(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase)
{
	dim3 block_size(8, 8);
	dim3 grid_size;
	grid_size.x = input_img.cols / block_size.x;
	grid_size.y = input_img.rows / block_size.y;

	if(input_img.cols % block_size.x)
	{
		grid_size.x++;
	}
	if(input_img.rows % block_size.y)
	{
		grid_size.y++;
	}

	float* input_img_ptr = input_img.ptr< float >(0);
	float* magnitude_ptr = magnitude.ptr< float >(0);
	float* phase_ptr = phase.ptr< float >(0);

	float* device_input_img;
	float* device_magnitude;
	float* device_phase;

	hipHostGetDevicePointer(&device_input_img, input_img_ptr, 0);
	hipHostGetDevicePointer(&device_magnitude, magnitude_ptr, 0);
	hipHostGetDevicePointer(&device_phase, phase_ptr, 0);

	gradient_kernel<<<grid_size, block_size>>>(device_input_img,
		device_magnitude, device_phase, input_img.rows, input_img.cols,
		input_img.step1(), magnitude.step1(), phase.step1());
	hipDeviceSynchronize();
}

void HogGPU::create_descriptor_impl(cv::Mat gradients,
	cv::Size block_size,
	int num_bins,
	cv::Mat& descriptor)
{
	int total_cells = block_size.width * block_size.height;
	descriptor.create(total_cells, num_bins, CV_32FC1);
	int top_row = 0, bottom_row = 0, left_col = 0, right_col = 0;
	int row_step = gradients.rows / block_size.height;
	int extra_rows = gradients.rows % block_size.height;
	int col_step = gradients.cols / block_size.width;
	int extra_cols = gradients.cols % block_size.width;

	for(int i = 0; i < block_size.height; ++i)
	{
		bottom_row = top_row += row_step;
		if(extra_rows > 0)
		{
			extra_rows--;
			bottom_row++;
		}
		cv::Mat temp = gradients.rowRange(top_row, bottom_row);
		for(int j = 0; j < block_size.width; ++j)
		{
			right_col = left_col += col_step;
			if(extra_cols > 0)
			{
				extra_cols--;
				right_col++;
			}
			calc_histogram(temp.colRange(left_col, right_col), num_bins,
				descriptor.row(i * block_size.height + j));
			left_col = right_col;
		}
		extra_cols = gradients.cols % block_size.width;
		top_row = bottom_row;
	}

	descriptor.reshape(1, 1);
}

void HogGPU::calc_histogram(cv::Mat gradients,
	int num_bins,
	cv::Mat histogram)
{
	float bin_size = 360.0f / (float)num_bins;
	histogram = cv::Mat(1, num_bins, CV_32FC1, 0.0f);

	//TODO Split more efficiently (maybe use reshape to get only one channel)
	cv::Mat aux[2];
	cv::split(gradients, aux);
	cv::Mat mag = aux[0];
	cv::Mat phase = aux[1];

	int left_bin, right_bin;
	float delta;

	float mag_total = 0.0f;

	for(int i = 0; i < mag.rows; ++i)
	{
		for(int j = 0; j < mag.cols; ++j)
		{
			if(mag.at< float >(i, j) > 0)
			{
				left_bin = (int)floor(
					(phase.at< float >(i, j) - bin_size / 2.0f) / bin_size);
				if(left_bin < 0)
					left_bin += num_bins;
				right_bin = (left_bin + 1) % num_bins;

				delta = (phase.at< float >(i, j) / bin_size) - right_bin;
				if(right_bin == 0)
					delta -= num_bins;

				histogram.at< float >(left_bin) += (0.5 - delta)
					* mag.at< float >(i, j);
				histogram.at< float >(right_bin) += (0.5 + delta)
					* mag.at< float >(i, j);
				mag_total += mag.at< float >(i, j);
			}
		}
	}

	for(int i = 0; i < num_bins; ++i)
	{
		histogram.at< float >(i) /= mag_total;
	}
}

} /* namespace lib */
} /* namespace ghog */

