#include "hip/hip_runtime.h"
/*
 * HogGPU.cpp
 *
 *  Created on: May 12, 2015
 *      Author: marcelo
 */

#include <include/HogGPU.h>
#include "HogGPU_impl.cuh"

#include <iostream>

#include <boost/thread.hpp>

#include <opencv2/gpu/gpu.hpp>

#include <include/Utils.h>

namespace ghog
{
namespace lib
{

HogGPU::HogGPU(std::string settings_file) :
	HogDescriptor(settings_file)
{

}

HogGPU::~HogGPU()
{
// TODO Auto-generated destructor stub
}

void HogGPU::alloc_buffer(cv::Size buffer_size,
	int type,
	cv::Mat& buffer)
{
	cv::gpu::CudaMem cudamem(buffer_size.height, buffer_size.width, type,
		cv::gpu::CudaMem::ALLOC_ZEROCOPY);
	buffer = cudamem.createMatHeader();
	buffer.refcount = cudamem.refcount;
	buffer.addref();
	buffer.setTo(0);
}

GHOG_LIB_STATUS HogGPU::image_normalization(cv::Mat& image,
	ImageCallback* callback)
{
	boost::thread(&HogGPU::image_normalization_async, this, image, callback)
		.detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::image_normalization_sync(cv::Mat& image)
{
	//TODO
}

GHOG_LIB_STATUS HogGPU::calc_gradient(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase,
	GradientCallback* callback)
{
	boost::thread(&HogGPU::calc_gradient_async, this, input_img, magnitude,
		phase, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::calc_gradient_sync(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase)
{
	dim3 block_size(8, 8);
	dim3 grid_size;
	grid_size.x = input_img.cols / block_size.x;
	grid_size.y = input_img.rows / block_size.y;

	if(input_img.cols % block_size.x)
	{
		grid_size.x++;
	}
	if(input_img.rows % block_size.y)
	{
		grid_size.y++;
	}

	float* input_img_ptr = input_img.ptr< float >(0);
	float* magnitude_ptr = magnitude.ptr< float >(0);
	float* phase_ptr = phase.ptr< float >(0);

	float* device_input_img;
	float* device_magnitude;
	float* device_phase;

	hipHostGetDevicePointer(&device_input_img, input_img_ptr, 0);
	hipHostGetDevicePointer(&device_magnitude, magnitude_ptr, 0);
	hipHostGetDevicePointer(&device_phase, phase_ptr, 0);

	gradient_kernel<<<grid_size, block_size>>>(device_input_img,
		device_magnitude, device_phase, input_img.rows, input_img.cols,
		input_img.step1(), magnitude.step1(), phase.step1());
	hipDeviceSynchronize();
}

void HogGPU::calc_histogram(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat histogram)
{
	float bin_size = 360.0f / (float)_num_bins;

	int left_bin, right_bin;
	float delta;

	float mag_total = 0.0f;

	for(int i = 0; i < magnitude.rows; ++i)
	{
		for(int j = 0; j < magnitude.cols; ++j)
		{
			if(magnitude.at< float >(i, j) > 0)
			{
				left_bin = (int)floor(
					(phase.at< float >(i, j) - bin_size / 2.0f) / bin_size);
				if(left_bin < 0)
					left_bin += _num_bins;
				right_bin = (left_bin + 1) % _num_bins;

				delta = (phase.at< float >(i, j) / bin_size) - right_bin;
				if(right_bin == 0)
					delta -= _num_bins;

				histogram.at< float >(left_bin) += (0.5 - delta)
					* magnitude.at< float >(i, j);
				histogram.at< float >(right_bin) += (0.5 + delta)
					* magnitude.at< float >(i, j);
				mag_total += magnitude.at< float >(i, j);
			}
		}
	}

	for(int i = 0; i < _num_bins; ++i)
	{
		histogram.at< float >(i) /= mag_total;
	}
}

void HogDescriptor::normalize_blocks(cv::Mat& descriptor)
{
	int cells_per_block = _block_size.height * _block_size.width;
	int elements_per_block = cells_per_block * _num_bins;

	for(int i = 0; i < descriptor.cols; i += elements_per_block)
	{
		float L1_norm = 0.0f;
		for(int j = 0; j < elements_per_block; ++j)
		{
			L1_norm += descriptor.at< float >(i + j);
		}
		for(int j = 0; j < elements_per_block; ++j)
		{
			descriptor.at< float >(i + j) = sqrt(
				descriptor.at< float >(i + j) / L1_norm);
		}
	}
}

//GHOG_LIB_STATUS HogGPU::classify(cv::Mat img,
//	ClassifyCallback* callback)
//{
//	boost::thread(&HogGPU::classify_async, this, img, callback).detach();
//	return GHOG_LIB_STATUS_OK;
//}
//
//bool HogGPU::classify_sync(cv::Mat img)
//{
//	bool ret = false;
//	cv::Mat resized;
//	image_normalization_sync(img);
//	cv::Mat grad_mag;
//	cv::Mat grad_phase;
//	calc_gradient_sync(img, grad_mag, grad_phase);
//	cv::Mat descriptor;
//	create_descriptor_sync(grad_mag, grad_phase, descriptor);
//	cv::Mat output = _classifier->classify_sync(descriptor);
//	if(output.at< float >(0) > 0)
//	{
//		ret = true;
//	}
//	return ret;
//}
//
//GHOG_LIB_STATUS HogGPU::locate(cv::Mat img,
//	cv::Rect roi,
//	cv::Size window_size,
//	cv::Size window_stride,
//	LocateCallback* callback)
//{
//	boost::thread(&HogGPU::locate_async, this, img, roi, window_size,
//		window_stride, callback).detach();
//	return GHOG_LIB_STATUS_OK;
//}
//
//std::vector< cv::Rect > HogGPU::locate_sync(cv::Mat img,
//	cv::Rect roi,
//	cv::Size window_size,
//	cv::Size window_stride)
//{
//	std::vector< cv::Rect > ret;
//	return ret;
//}

} /* namespace lib */
} /* namespace ghog */

