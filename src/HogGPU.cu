/*
 * HogGPU.cpp
 *
 *  Created on: May 12, 2015
 *      Author: marcelo
 */

#include <include/HogGPU.h>
#include "HogGPU_impl.cuh"

#include <iostream>

#include <boost/thread.hpp>

#include <opencv2/gpu/gpu.hpp>

#include <include/Utils.h>

namespace ghog
{
namespace lib
{

HogGPU::HogGPU(std::string settings_file) :
	HogDescriptor(settings_file)
{

}

HogGPU::~HogGPU()
{
// TODO Auto-generated destructor stub
}

void HogGPU::alloc_buffer(cv::Size buffer_size,
	int type,
	cv::Mat& buffer,
	int padding_size)
{
	cv::gpu::CudaMem cudamem(buffer_size.height + 2 * padding_size,
		buffer_size.width + 2 * padding_size, type,
		cv::gpu::CudaMem::ALLOC_ZEROCOPY);
	buffer = cudamem.createMatHeader().rowRange(padding_size,
		cudamem.rows - padding_size).colRange(padding_size,
		cudamem.cols - padding_size);
	buffer.refcount = cudamem.refcount;
	buffer.addref();
	cv::Mat header_temp = cudamem.createMatHeader();
	header_temp.setTo(0);
}

GHOG_LIB_STATUS HogGPU::image_normalization(cv::Mat& image,
	ImageCallback* callback)
{
	boost::thread(&HogGPU::image_normalization_async, this, image, callback)
		.detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::image_normalization_sync(cv::Mat& image)
{
	dim3 block_size(3, 64, 1);
	dim3 grid_size;
	grid_size.x = image.cols / block_size.y;
	grid_size.y = image.rows / block_size.z;

	if(image.cols % block_size.y)
	{
		grid_size.x++;
	}
	if(image.rows % block_size.z)
	{
		grid_size.y++;
	}

	float* input_img_ptr = image.ptr< float >(0);
	float* device_input_img;
	hipHostGetDevicePointer(&device_input_img, input_img_ptr, 0);

	gamma_norm_kernel<<<grid_size, block_size>>>(device_input_img, image.rows,
		image.cols, image.step1());
	hipDeviceSynchronize();
}

GHOG_LIB_STATUS HogGPU::calc_gradient(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase,
	GradientCallback* callback)
{
	boost::thread(&HogGPU::calc_gradient_async, this, input_img, magnitude,
		phase, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::calc_gradient_sync(cv::Mat input_img,
	cv::Mat& magnitude,
	cv::Mat& phase)
{
	dim3 block_size(3, 64, 1);
	dim3 grid_size;
	grid_size.x = input_img.cols / block_size.y;
	grid_size.y = input_img.rows / block_size.z;

	if(input_img.cols % block_size.y)
	{
		grid_size.x++;
	}
	if(input_img.rows % block_size.z)
	{
		grid_size.y++;
	}

	float* input_img_ptr = input_img.ptr< float >(0);
	float* magnitude_ptr = magnitude.ptr< float >(0);
	float* phase_ptr = phase.ptr< float >(0);

	float* device_input_img;
	float* device_magnitude;
	float* device_phase;

	hipHostGetDevicePointer(&device_input_img, input_img_ptr, 0);
	hipHostGetDevicePointer(&device_magnitude, magnitude_ptr, 0);
	hipHostGetDevicePointer(&device_phase, phase_ptr, 0);

	gradient_kernel<<<grid_size, block_size>>>(device_input_img,
		device_magnitude, device_phase, input_img.rows, input_img.cols,
		input_img.step1(), magnitude.step1(), phase.step1());
	hipDeviceSynchronize();
}

GHOG_LIB_STATUS HogGPU::create_descriptor(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat& descriptor,
	DescriptorCallback* callback)
{
	boost::thread(&HogGPU::create_descriptor_async, this, magnitude, phase,
		descriptor, callback).detach();
	return GHOG_LIB_STATUS_OK;
}

void HogGPU::create_descriptor_sync(cv::Mat magnitude,
	cv::Mat phase,
	cv::Mat& descriptor,
	cv::Mat& histograms)
{
	//TODO: verify that magnitude and phase have correct size and type.
	//TODO: verify that the descriptor has correct size and type
	//TODO: possibly preallocate histograms auxiliary matrix

	cv::Size hog_block_grid(
		((_cell_grid.width - _block_size.width) / _block_stride.width) + 1,
		((_cell_grid.height - _block_size.height) / _block_stride.height) + 1);

	dim3 block_size_hist(64, 1);
	dim3 grid_size;
	grid_size.x = magnitude.cols / block_size_hist.x;
	grid_size.y = _cell_grid.height / block_size_hist.y;

	if(magnitude.cols % block_size_hist.x)
	{
		grid_size.x++;
	}
	if(_cell_grid.height % block_size_hist.y)
	{
		grid_size.y++;
	}

	float* magnitude_ptr = magnitude.ptr< float >(0);
	float* phase_ptr = phase.ptr< float >(0);
	float* descriptor_ptr = descriptor.ptr< float >(0);
	float* histograms_ptr = histograms.ptr< float >(0);

	float* device_magnitude;
	float* device_phase;
	float* device_descriptor;
	float* device_histograms;

	hipHostGetDevicePointer(&device_magnitude, magnitude_ptr, 0);
	hipHostGetDevicePointer(&device_phase, phase_ptr, 0);
	hipHostGetDevicePointer(&device_descriptor, descriptor_ptr, 0);
	hipHostGetDevicePointer(&device_histograms, histograms_ptr, 0);

//	int cell_row_step = _cell_grid.width * _num_bins;

	histogram_kernel<<<grid_size, block_size_hist>>>(device_magnitude,
		device_phase, device_histograms, magnitude.cols, magnitude.rows,
		_cell_grid.width, _cell_grid.height, magnitude.step1(), phase.step1(),
		histograms.step1(), _cell_size.width, _cell_size.height, _num_bins);

	dim3 block_size_norm(9, 4, 8);

	grid_size.x = hog_block_grid.width / 8; // 8 hog blocks per thread block
	grid_size.y = hog_block_grid.height;

	if(hog_block_grid.width % 8)
	{
		grid_size.x++;
	}

	hipDeviceSynchronize();

	block_normalization_kernel<<<grid_size, block_size_norm>>>(
		device_histograms, device_descriptor, histograms.step1(),
		hog_block_grid.width, hog_block_grid.height, _block_size.width,
		_block_size.height, _num_bins, _cell_grid.width, _block_stride.width,
		_block_stride.height);
	hipDeviceSynchronize();
}

} /* namespace lib */
} /* namespace ghog */

