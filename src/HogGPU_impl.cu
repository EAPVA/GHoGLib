#include "hip/hip_runtime.h"
#include "HogGPU_impl.cuh"

#include "hip/hip_math_constants.h"

__global__ void gamma_norm_kernel(float* img,
	int image_height,
	int image_width,
	int image_step)
{
	int channel = threadIdx.x;
	int pixel_x = blockIdx.x * blockDim.y + threadIdx.y;
	if(pixel_x >= image_width)
	{
		return;
	}
	int pixel_y = blockIdx.y * blockDim.z + threadIdx.z;
	if(pixel_y >= image_height)
	{
		return;
	}

	int in_pixel_idx = pixel_y * image_step + pixel_x * 3 + channel;

	img[in_pixel_idx] = sqrt(img[in_pixel_idx] / 256.0f);

}

__global__ void gradient_kernel(float* input_img,
	float* magnitude,
	float* phase,
	int image_height,
	int image_width,
	int input_image_step,
	int magnitude_step,
	int phase_step)
{
	__shared__ float s_magnitude[192];
	__shared__ float s_phase[192];

	int channel = threadIdx.x;
	int pixel_x = blockIdx.x * blockDim.y + threadIdx.y;
	if(pixel_x >= image_width)
	{
		return;
	}
	int pixel_y = blockIdx.y * blockDim.z + threadIdx.z;
	if(pixel_y >= image_height)
	{
		return;
	}

	int bs_x = threadIdx.y;
	int bs_y = threadIdx.z;
	int bs_step = 3 * blockDim.y;
	int bs_idx = bs_y * bs_step + bs_x * 3 + channel;

	int in_pixel_idx = pixel_y * input_image_step + pixel_x * 3 + channel;
	int mag_pixel_idx = pixel_y * magnitude_step + pixel_x;
	int phase_pixel_idx = pixel_y * phase_step + pixel_x;

	float dx = input_img[in_pixel_idx + 3];
	dx -= input_img[in_pixel_idx - 3];
	float dy = input_img[in_pixel_idx + input_image_step];
	dy -= input_img[in_pixel_idx - input_image_step];

	s_magnitude[bs_idx] = sqrt(dx * dx + dy * dy);
	s_phase[bs_idx] = (atan2(dy, dx) + HIP_PI_F) / (2.0f * HIP_PI_F);

	__syncthreads();

	if(channel == 0)
	{
		float mag_max = s_magnitude[3 * threadIdx.y];
		int k = 0;
		if(s_magnitude[3 * threadIdx.y + 1] > mag_max)
		{
			mag_max = s_magnitude[3 * threadIdx.y + 1];
			k = 1;
		}
		if(s_magnitude[3 * threadIdx.y + 2] > mag_max)
		{
			mag_max = s_magnitude[3 * threadIdx.y + 1];
			k = 2;
		}

		magnitude[mag_pixel_idx] = mag_max;
		phase[phase_pixel_idx] = s_phase[3 * threadIdx.y + k];
	}
}

__global__ void histogram_kernel(float* magnitude,
	float* phase,
	float* histograms,
	int input_width,
	int input_height,
	int cell_grid_width,
	int cell_grid_height,
	int magnitude_step,
	int phase_step,
	int histograms_step,
	int cell_width,
	int cell_height,
	int num_bins)
{
	__shared__ int s_lbin_pos[64];
	__shared__ float s_lbin[64];
	__shared__ int s_rbin_pos[64];
	__shared__ float s_rbin[64];
	__shared__ float s_hist[9 * 2];
	__shared__ float s_hist_total[2];

	if(threadIdx.x < 18)
	{
		s_hist[threadIdx.x] = 0.0f;
	}
	if(threadIdx.x < 2)
	{
		s_hist_total[threadIdx.x] = 0.0f;
	}

	int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
	if(pixel_x >= input_width)
	{
		return;
	}
	int pixel_y = 32 * (blockIdx.y * blockDim.y + threadIdx.y);
	if(pixel_y >= input_height)
	{
		return;
	}
	int cell_y = pixel_y / cell_height;
	int cell_x = pixel_x / cell_width;

	for(int i = 0; i < 32; ++i)
	{
		int mag_pixel_idx = pixel_y * magnitude_step + pixel_x;
		int phase_pixel_idx = pixel_y * phase_step + pixel_x;

		float bin_size = 1.0f / (float)num_bins;
		int left_bin = (int)floor(
			(phase[phase_pixel_idx] - bin_size / 2.0f) / bin_size);
		left_bin = (left_bin + num_bins) % num_bins;
		//Might be outside the range. First use on the formula below, then fix the range.
		int right_bin = (left_bin + 1);
		float delta = (phase[phase_pixel_idx] / bin_size) - right_bin;
		if(delta < -0.5)
		{
			delta += num_bins;
		}
		//Fix range for right_bin
		right_bin = right_bin % num_bins;

		s_lbin_pos[threadIdx.x] = left_bin;
		s_lbin[threadIdx.x] = (0.5 - delta) * magnitude[mag_pixel_idx];
		s_rbin_pos[threadIdx.x] = right_bin;
		s_lbin[threadIdx.x] = (0.5 + delta) * magnitude[mag_pixel_idx];

//	s_hist[threadIdx.x] = 0.0f;

		__syncthreads();

		if(threadIdx.x < 2)
		{
			int s_hist_idx = 9 * threadIdx.x;
			for(int i = 0; i < 32; ++i)
			{
				s_hist[s_hist_idx + s_lbin_pos[32 * threadIdx.x + i]] +=
					s_lbin[32 * threadIdx.x + i];
				s_hist[s_hist_idx + s_rbin_pos[32 * threadIdx.x + i]] +=
					s_rbin[32 * threadIdx.x + i];
				s_hist_total[threadIdx.x] += s_lbin[32 * threadIdx.x + i]
					+ s_rbin[32 * threadIdx.x + i];
			}
		}
		pixel_y++;

		__syncthreads();
	}

	int cell_pos = threadIdx.x / 9;
	int out_idx = histograms_step * cell_y + num_bins * (cell_x) + threadIdx.x;
//	return;

	if(threadIdx.x < 18)
	{
		if(s_hist_total[cell_pos] > 0.1)
		{
			s_hist[threadIdx.x] /= s_hist_total[cell_pos];
		}
		histograms[out_idx] = s_hist[threadIdx.x];
	}
}

__global__ void block_normalization_kernel(float* histograms,
	float* descriptor,
	int histograms_step,
	int block_grid_width,
	int block_grid_height,
	int block_width,
	int block_height,
	int num_bins,
	int cell_grid_width,
	int block_stride_x,
	int block_stride_y)
{
	//Each thread block will process 8 hog blocks.
	__shared__ float s_blocks[9 * 4 * 8];
	__shared__ float L1_norm[8];
	int block_x = blockIdx.x * 8 + threadIdx.z;
	if(block_x >= block_grid_width)
	{
		return;
	}
	int block_y = blockIdx.y;
	if(block_y >= block_grid_height)
	{
		return;
	}
	int block_idx = block_y * blockDim.y + block_x;
	int cell_x = block_x * block_stride_x + threadIdx.y % 2;
	int cell_y = block_y * block_stride_y + threadIdx.y / 2;
	int hist_idx = histograms_step * cell_y + num_bins * (cell_x) + threadIdx.x;

	int s_blocks_idx = 9 * threadIdx.y + threadIdx.x;
	s_blocks[s_blocks_idx] = histograms[hist_idx];

	__syncthreads();

	int thread_id = 36 * threadIdx.z + 9 * threadIdx.y + threadIdx.x;
	int elements_per_block = block_height * block_width * num_bins;
	if(thread_id < 8)
	{
		L1_norm[thread_id] = 0.0f;
		for(int i = 0; i < elements_per_block; ++i)
		{
			L1_norm[thread_id] += s_blocks[elements_per_block * thread_id + i];
		}
	}

	__syncthreads();

	descriptor[elements_per_block * block_idx + s_blocks_idx] =
		s_blocks[s_blocks_idx];
}
