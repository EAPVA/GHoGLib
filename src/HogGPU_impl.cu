#include "hip/hip_runtime.h"
#include "HogGPU_impl.cuh"

#include "hip/hip_math_constants.h"

namespace ghog
{
namespace lib
{
namespace gpu
{

__global__ void gamma_norm_kernel(float* img,
	int image_height,
	int image_width,
	int image_step)
{
	// The thread block has size (3,n). The first dimension of the thread block
	// corresponds to color channels.
	int channel = threadIdx.x;
	// The columns of the image are mapped to the first dimension of the block
	// grid, but to the second dimension of the thread block, as the first
	// already corresponds to color channels.
	int pixel_x = blockIdx.x * blockDim.y + threadIdx.y;
	// If current position is outside the image, stop here
	if(pixel_x >= image_width)
	{
		return;
	}
	// The columns of the image are mapped to the second dimension of the block
	// grid, but to the third dimension of the thread block.
	int pixel_y = blockIdx.y * blockDim.z + threadIdx.z;
	// If current position is outside the image, stop here
	if(pixel_y >= image_height)
	{
		return;
	}

	// Each row has image_step pixels and each pixel has three channels
	int in_pixel_idx = pixel_y * image_step + pixel_x * 3 + channel;

	// Finally perform the normalization
	img[in_pixel_idx] = sqrt(img[in_pixel_idx] / 256.0f);

}

__global__ void gradient_kernel(float* input_img,
	float* magnitude,
	float* phase,
	int image_height,
	int image_width,
	int input_image_step,
	int magnitude_step,
	int phase_step)
{
	//TODO: make the buffer sizes dependent on an input or template parameter.
	//Each thread block needs to store 2 * 64 bytes * 3 floats per channel = 2 * 192
	__shared__ float s_magnitude[192];
	__shared__ float s_phase[192];

	// The thread block has size (3,n). The first dimension of the thread block
	// corresponds to color channels.
	int channel = threadIdx.x;
	// The columns of the image are mapped to the first dimension of the block
	// grid, but to the second dimension of the thread block, as the first
	// already corresponds to color channels.
	int pixel_x = blockIdx.x * blockDim.y + threadIdx.y;
	// If current position is outside the image, stop here
	if(pixel_x >= image_width)
	{
		return;
	}
	// The columns of the image are mapped to the second dimension of the block
	// grid, but to the third dimension of the thread block.
	int pixel_y = blockIdx.y * blockDim.z + threadIdx.z;
	// If current position is outside the image, stop here
	if(pixel_y >= image_height)
	{
		return;
	}

	//The indexes for the internal buffer don't depend on the block index.
	int bs_x = threadIdx.y;
	int bs_y = threadIdx.z;
	int bs_step = 3 * blockDim.y;
	int bs_idx = bs_y * bs_step + bs_x * 3 + channel;

	// Each row has input_image_step size and each pixel has three channels
	int in_pixel_idx = pixel_y * input_image_step + pixel_x * 3 + channel;
	// Each row has magnitude_step size
	int mag_pixel_idx = pixel_y * magnitude_step + pixel_x;
	// Each row has phase_step size
	int phase_pixel_idx = pixel_y * phase_step + pixel_x;

	// Calculate the X and Y coordinates of the gradient.
	float dx = input_img[in_pixel_idx + 3];
	dx -= input_img[in_pixel_idx - 3];
	float dy = input_img[in_pixel_idx + input_image_step];
	dy -= input_img[in_pixel_idx - input_image_step];

	// Store the magnitude and the phase of the gradient on the shared buffer.
	s_magnitude[bs_idx] = sqrt(dx * dx + dy * dy);
	// Normalize the phase output to [0,1] rotations.
	s_phase[bs_idx] = (atan2(dy, dx) + HIP_PI_F) / (2.0f * HIP_PI_F);

	// Wait until all threads finish this step.
	__syncthreads();

	//Only one each three threads will verify the max value and store the result.
	if(channel == 0)
	{
		float mag_max = s_magnitude[3 * threadIdx.y];
		int k = 0;
		if(s_magnitude[3 * threadIdx.y + 1] > mag_max)
		{
			mag_max = s_magnitude[3 * threadIdx.y + 1];
			k = 1;
		}
		if(s_magnitude[3 * threadIdx.y + 2] > mag_max)
		{
			mag_max = s_magnitude[3 * threadIdx.y + 2];
			k = 2;
		}

		magnitude[mag_pixel_idx] = mag_max;
		phase[phase_pixel_idx] = s_phase[3 * threadIdx.y + k];
	}
}

__global__ void histogram_kernel(float* magnitude,
	float* phase,
	float* histograms,
	int input_width,
	int input_height,
	int cell_grid_width,
	int cell_grid_height,
	int magnitude_step,
	int phase_step,
	int histograms_step,
	int cell_width,
	int cell_height,
	int num_bins)
{
	//TODO: make the buffer sizes dependent on an input or template parameter.
	// Each thread block needs to store intermediate results for 64 gradients
	// and also 8 different histograms, each with 9 bins.
	__shared__ int s_lbin_pos[64];
	__shared__ float s_lbin[64];
	__shared__ int s_rbin_pos[64];
	__shared__ float s_rbin[64];
	__shared__ float s_hist[9 * 8];

	// The columns of the image are mapped to the first dimension of the block
	// grid and the first dimension of the thread block.
	int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
	// If current position is outside the image, stop here
	if(pixel_x >= input_width)
	{
		return;
	}
	// The columns of the image are mapped to the second dimension of the block
	// grid and the second dimension of the thread block.
	int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
	// If current position is outside the image, stop here
	if(pixel_y >= input_height)
	{
		return;
	}

	// Each row has magnitude_step size
	int mag_pixel_idx = pixel_y * magnitude_step + pixel_x;
	// Each row has phase_step size
	int phase_pixel_idx = pixel_y * phase_step + pixel_x;

	// The phase was previously normalized to [0,1]
	float bin_size = 1.0f / (float)num_bins;
	// By dividing by the bin size and taking the integer part, you find out
	// inside which bin the gradient is at. If it's greater than the middle of the bin
	// it will be divided between this one and the next, if it's lesser it will
	// be divided between this and the previous one. By subtracting 0.5 before
	// taking the integer part, the division will always be between this bin and
	// the next.
	int left_bin = (int)floor((phase[phase_pixel_idx] / bin_size) - 0.5f);
	// The result of the previous operation might be negative. If so, the next
	// bit fixes that. Otherwise that changes nothing.
	left_bin = (left_bin + num_bins) % num_bins;
	// Take the next bin as the right bin.
	// If the left bin is the last one, this will be outside range. Wait a bit
	// before taking the remainder, because this value needs to be used in the
	// formula below.
	int right_bin = (left_bin + 1);
	// Calculate the distance between the gradient phase and the limit between
	// the left and right bins. Normalized by the bin size, the limit is equal
	// to the right bin identifier.
	float delta = (phase[phase_pixel_idx] / bin_size) - right_bin;
	if(delta < -0.5)
	{
		delta += num_bins;
	}
	//Fix range for right_bin now
	right_bin = right_bin % num_bins;

	// Store the bin positions and amounts for each bin on shared buffers.
	s_lbin_pos[threadIdx.x] = left_bin;
	s_lbin[threadIdx.x] = (0.5 - delta) * magnitude[mag_pixel_idx];
	s_rbin_pos[threadIdx.x] = right_bin;
	s_rbin[threadIdx.x] = (0.5 + delta) * magnitude[mag_pixel_idx];

	// Wait for other threads.
	__syncthreads();

	// Initialize histograms shared buffer.
	s_hist[threadIdx.x] = 0.0f;
	if(threadIdx.x < 8)
	{
		s_hist[threadIdx.x + 64] = 0.0f;
	}

	int cell_y = pixel_y / cell_height;

	// Each partial histogram will be calculated by only one thread.
	if(threadIdx.x < 8)
	{
		int s_hist_idx = 9 * threadIdx.x;
		for(int i = 1; i < 8; ++i)
		{
			s_hist[s_hist_idx + s_lbin_pos[8 * threadIdx.x + i]] += s_lbin[8
				* threadIdx.x + i];
			s_hist[s_hist_idx + s_rbin_pos[8 * threadIdx.x + i]] += s_rbin[8
				* threadIdx.x + i];
		}
	}

	// Wait until all threads finish.
	__syncthreads();

	// Add to the complete histogram sum using atomic operations.
	int out_idx = cell_y * histograms_step + threadIdx.x;
	atomicAdd(&(histograms[out_idx]), s_hist[threadIdx.x]);

	if(threadIdx.x < 8)
	{
		atomicAdd(&(histograms[out_idx + 64]), s_hist[threadIdx.x + 64]);
	}
}

__global__ void block_normalization_kernel(float* histograms,
	float* descriptor,
	int histograms_step,
	int block_grid_width,
	int block_grid_height,
	int block_width,
	int block_height,
	int num_bins,
	int cell_grid_width,
	int block_stride_x,
	int block_stride_y)
{
	//TODO: make the buffer sizes dependent on an input or template parameter.
	// Each thread block will process 8 hog blocks. Each hog block has 4 cells.
	// Each cell has 9 bins.
	__shared__ float s_blocks[9 * 4 * 8];
	__shared__ float L1_norm[8];
	int block_x = blockIdx.x * 8 + threadIdx.z;
	if(block_x >= block_grid_width)
	{
		return;
	}
	int block_y = blockIdx.y;
	if(block_y >= block_grid_height)
	{
		return;
	}
	int block_idx = block_y * blockDim.y + block_x;
	int cell_x = block_x * block_stride_x + threadIdx.y % 2;
	int cell_y = block_y * block_stride_y + threadIdx.y / 2;
	int hist_idx = histograms_step * cell_y + num_bins * (cell_x) + threadIdx.x;

	int s_blocks_idx = 9 * threadIdx.y + threadIdx.x;
	s_blocks[s_blocks_idx] = histograms[hist_idx];

	__syncthreads();

	int thread_id = 36 * threadIdx.z + 9 * threadIdx.y + threadIdx.x;
	int elements_per_block = block_height * block_width * num_bins;
	if(thread_id < 8)
	{
		L1_norm[thread_id] = 0.0f;
		for(int i = 0; i < elements_per_block; ++i)
		{
			L1_norm[thread_id] += s_blocks[elements_per_block * thread_id + i];
		}
	}

	__syncthreads();

	descriptor[elements_per_block * block_idx + s_blocks_idx] =
		s_blocks[s_blocks_idx] / L1_norm[threadIdx.z];
}

} /* namespace gpu */
} /* namespace ghog */
} /* namespace lib */
