#include "hip/hip_runtime.h"
#include "HogGPU_impl.cuh"

__global__ void gradient_kernel(float* input_img,
	float* magnitude,
	float* phase,
	int image_height,
	int image_width,
	int input_image_step,
	int magnitude_step,
	int phase_step)
{
	int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
	if(pixel_x >= image_width)
	{
		return;
	}
	int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
	if(pixel_y >= image_height)
	{
		return;
	}

	int in_pixel_idx = pixel_y * input_image_step + pixel_x * 3;
	int mag_pixel_idx = pixel_y * magnitude_step + pixel_x;
	int phase_pixel_idx = pixel_y * phase_step + pixel_x;

	float dx, dy;
	float mag_max = 0.0f;
	float phase_max = 0.0f;

	for(int i = 0; i < 3; ++i)
	{
		dx = input_img[in_pixel_idx + 3] - input_img[in_pixel_idx - 3];
		dy = input_img[in_pixel_idx + input_image_step]
			- input_img[in_pixel_idx - input_image_step];
		float mag = sqrt(dx * dx + dy * dy);;

		if (mag > mag_max) {
			phase_max = atan2(dy, dx);
		}
	}

	magnitude[mag_pixel_idx] = mag_max;
	phase[phase_pixel_idx] = phase_max;
}

__global__ void histogram_kernel(float* magnitude,
	float* phase,
	float* histograms,
	int input_width,
	int input_height,
	int magnitude_step,
	int phase_step,
	int cell_row_step,
	int cell_width,
	int cell_height,
	int num_bins)
{
	int cell_x = blockIdx.x * blockDim.x + threadIdx.x;
	if(cell_x >= input_width)
	{
		return;
	}
	int cell_y = blockIdx.y * blockDim.y + threadIdx.y;
	if(cell_y >= input_height)
	{
		return;
	}

	int left_bin, right_bin;
	int pixel_x = cell_x * cell_width;
	int pixel_y = cell_y * cell_height;
	int mag_pixel_idx;
	int phase_pixel_idx;
	int out_idx = cell_y * cell_row_step + cell_x * num_bins;
	int i, j;

	float delta = 0.0f;
	float bin_size = 360.0f / (float)num_bins;
	float mag_total = 0;

	for(i = 0; i < cell_height; ++i)
	{
		mag_pixel_idx = (pixel_y + i) * magnitude_step + pixel_x;
		phase_pixel_idx = (pixel_y + i) * phase_step + pixel_x;
		for(j = 0; j < cell_width; ++j)
		{
			left_bin = (int)floor(
				(phase[phase_pixel_idx + j] - bin_size / 2.0f) / bin_size);
			left_bin = (left_bin + num_bins) % num_bins;
			//Might be outside the range. First use on the formula below, then fix the range.
			right_bin = (left_bin + 1);

			delta = (phase[phase_pixel_idx + j] / bin_size) - right_bin;

			//Fix range for right_bin
			right_bin = right_bin % num_bins;

			histograms[out_idx + left_bin] += (0.5 - delta)
				* magnitude[mag_pixel_idx + j];
			histograms[out_idx + right_bin] += (0.5 + delta)
				* magnitude[mag_pixel_idx + j];
			mag_total += magnitude[mag_pixel_idx + j];
		}
	}

	for(i = 0; i < num_bins; ++i)
	{
		histograms[out_idx + i] /= mag_total;
	}
}

__global__ void block_normalization_kernel(float* histograms,
	float* descriptor,
	int block_grid_width,
	int block_grid_height,
	int block_width,
	int block_height,
	int num_bins,
	int cell_grid_width,
	int block_stride_x,
	int block_stride_y)
{
	int block_x = blockIdx.x * blockDim.x + threadIdx.x;
	if(block_x >= block_grid_width)
	{
		return;
	}
	int block_y = blockIdx.y * blockDim.y + threadIdx.y;
	if(block_y >= block_grid_height)
	{
		return;
	}
	int block_idx = block_y * block_grid_width + block_x;
	int elements_per_block = block_width * block_height * num_bins;
	int block_pos = block_idx * elements_per_block;
	int block_pos_delta = 0;

	int cell_x = block_x * block_stride_x;
	int cell_y = block_y * block_stride_y;
	int cell_idx;
	int hist_pos;
	int i, j, k;

	float L1_norm = 0.0f;

	for(i = 0; i < block_height; ++i)
	{
		cell_idx = ((cell_y + i) * cell_grid_width) + cell_x;
		for(j = 0; j < block_width; ++j)
		{
			hist_pos = (cell_idx + j) * num_bins;
			for(k = 0; k < num_bins; ++k)
			{
				L1_norm += histograms[hist_pos + k];
				descriptor[block_pos + block_pos_delta] = histograms[hist_pos
					+ k];
				block_pos_delta++;
			}
		}
	}

	for(i = 0; i < elements_per_block; ++i)
	{
		descriptor[block_pos + i] /= L1_norm;
	}
}
