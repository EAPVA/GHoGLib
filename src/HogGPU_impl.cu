#include "hip/hip_runtime.h"
#include "HogGPU_impl.cuh"

__global__ void gradient_kernel(float* input_img,
	float* magnitude,
	float* phase,
	int image_height,
	int image_width,
	int input_image_step,
	int magnitude_step,
	int phase_step)
{
	int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
	if(pixel_x >= image_width)
	{
		return;
	}
	int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
	if(pixel_y >= image_height)
	{
		return;
	}

	int in_pixel_idx = pixel_y * input_image_step + pixel_x;
	int mag_pixel_idx = pixel_y * magnitude_step + pixel_x;
	int phase_pixel_idx = pixel_y * phase_step + pixel_x;

	float dx = input_img[in_pixel_idx + 1] - input_img[in_pixel_idx - 1];
	float dy = input_img[in_pixel_idx + input_image_step]
		- input_img[in_pixel_idx - input_image_step];

	magnitude[mag_pixel_idx] = sqrt(dx * dx + dy * dy);
	phase[phase_pixel_idx] = atan2(dy, dx);
}
